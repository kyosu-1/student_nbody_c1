#include "hip/hip_runtime.h"
#include <chrono>
#include <hiprand/hiprand_kernel.h>

#include "allocator_config.h"
#include "../configuration.h"

#ifdef OPTION_RENDER
#include "../rendering.h"
#endif  // OPTION_RENDER


// Pre-declare all classes.
class Body;

using AllocatorT = SoaAllocator<kNumObjects, Body>;

class Body : public AllocatorT::Base {
 public:
  declare_field_types(
      Body,
      Body*,          // merge_target_
      float,          // pos_x_
      float,          // pos_y_,
      float,          // vel_x_,
      float,          // vel_y_,
      float,          // force_x_,
      float,          // force_y_,
      float,          // mass_
      bool,           // has_incoming_merge_
      bool)           // successful_merge_

 private:
  Field<Body, 0> merge_target_;
  Field<Body, 1> pos_x_;
  Field<Body, 2> pos_y_;
  Field<Body, 3> vel_x_;
  Field<Body, 4> vel_y_;
  Field<Body, 5> force_x_;
  Field<Body, 6> force_y_;
  Field<Body, 7> mass_;
  Field<Body, 8> has_incoming_merge_;
  Field<Body, 9> successful_merge_;

 public:
  __device__ Body(float pos_x, float pos_y, float vel_x, float vel_y, float mass);

  __device__ Body(int index);

  __device__ void compute_force();

  __device__ void apply_force(Body* other);

  __device__ void update();

  __device__ void check_merge_into_this(Body* other);

  __device__ void initialize_merge();

  __device__ void prepare_merge();

  __device__ void update_merge();

  __device__ void delete_merged();

  // Only for rendering and checksum computation.
  __device__ __host__ float pos_x() const { return pos_x_; }
  __device__ __host__ float pos_y() const { return pos_y_; }
  __device__ __host__ float vel_x() const { return vel_x_; }
  __device__ __host__ float vel_y() const { return vel_y_; }
  __device__ __host__ float mass() const { return mass_; }
};


// Allocator handles.
AllocatorHandle<AllocatorT>* allocator_handle;
__device__ AllocatorT* device_allocator;



__device__ Body::Body(float pos_x, float pos_y,
                      float vel_x, float vel_y, float mass)
    : pos_x_(pos_x), pos_y_(pos_y),
      vel_x_(vel_x), vel_y_(vel_y), mass_(mass) {}


__device__ void Body::compute_force() {
  force_x_ = 0.0f;
  force_y_ = 0.0f;
  device_allocator->device_do<Body>(&Body::apply_force, this);
}


__device__ Body::Body(int idx) {
  hiprandState rand_state;
  hiprand_init(kSeed, idx, 0, &rand_state);

  pos_x_ = 2 * hiprand_uniform(&rand_state) - 1;
  pos_y_ = 2 * hiprand_uniform(&rand_state) - 1;
  vel_x_ = (hiprand_uniform(&rand_state) - 0.5) / 1000;
  vel_y_ = (hiprand_uniform(&rand_state) - 0.5) / 1000;
  mass_ = (hiprand_uniform(&rand_state)/2 + 0.5) * kMaxMass;
}


__device__ void Body::apply_force(Body* other) {
  // Update `other`.
  if (other != this) {
    float dx = pos_x_ - other->pos_x_;
    float dy = pos_y_ - other->pos_y_;
    float dist = sqrt(dx*dx + dy*dy);
    float F = kGravityConstant * mass_ * other->mass_
        / (dist * dist + kDampeningFactor);
    other->force_x_ += F*dx / dist;
    other->force_y_ += F*dy / dist;
  }
}


__device__ void Body::update() {
  vel_x_ += force_x_*kTimeInterval / mass_;
  vel_y_ += force_y_*kTimeInterval / mass_;
  pos_x_ += vel_x_*kTimeInterval;
  pos_y_ += vel_y_*kTimeInterval;

  if (pos_x_ < -1 || pos_x_ > 1) {
    vel_x_ = -vel_x_;
  }

  if (pos_y_ < -1 || pos_y_ > 1) {
    vel_y_ = -vel_y_;
  }
}


__device__ void Body::check_merge_into_this(Body* other) {
  // Only merge into larger body.
  if (!other->has_incoming_merge_ && mass_ > other->mass_) {
    float dx = pos_x_ - other->pos_x_;
    float dy = pos_y_ - other->pos_y_;
    float dist_square = dx*dx + dy*dy;

    if (dist_square < kMergeThreshold*kMergeThreshold) {
      // Try to merge this one.
      // There is a race condition here: Multiple threads may try to merge
      // this body. Only one can win. That's OK.
      this->merge_target_ = other;
      other->has_incoming_merge_ = true;
    }
  }
}


__device__ void Body::initialize_merge() {
  merge_target_ = nullptr;
  has_incoming_merge_ = false;
  successful_merge_ = false;
}


__device__ void Body::prepare_merge() {
  device_allocator->template device_do<Body>(&Body::check_merge_into_this,
                                             this);
}


__device__ void Body::update_merge() {
  Body* m = merge_target_;
  if (m != nullptr) {
    if (m->merge_target_ == nullptr) {
      // Perform merge.
      float new_mass = mass_ + m->mass_;
      float new_vel_x = (vel_x_*mass_ + m->vel_x_*m->mass_) / new_mass;
      float new_vel_y = (vel_y_*mass_ + m->vel_y_*m->mass_) / new_mass;
      m->mass_ = new_mass;
      m->vel_x_ = new_vel_x;
      m->vel_y_ = new_vel_y;
      m->pos_x_ = (pos_x_ + m->pos_x_) / 2;
      m->pos_y_ = (pos_y_ + m->pos_y_) / 2;

      successful_merge_ = true;
    }
  }
}


__device__ void Body::delete_merged() {
  if (successful_merge_) {
    destroy(device_allocator, this);
  }
}


__global__ void kernel_initialize_bodies() {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  hiprandState rand_state;
  hiprand_init(kSeed, tid, 0, &rand_state);

  for (int i = tid; i < kNumBodies; i += blockDim.x * gridDim.x) {
    new(device_allocator) Body(
        /*pos_x=*/ 2 * hiprand_uniform(&rand_state) - 1,
        /*pos_y=*/ 2 * hiprand_uniform(&rand_state) - 1,
        /*vel_x=*/ (hiprand_uniform(&rand_state) - 0.5) / 1000,
        /*vel_y=*/ (hiprand_uniform(&rand_state) - 0.5) / 1000,
        /*mass=*/ (hiprand_uniform(&rand_state)/2 + 0.5) * kMaxMass);
  }
}


int checksum() {
  int result = 0;

  allocator_handle->template device_do<Body>([&](Body* body) {
    int Body_checksum = static_cast<int>((body->pos_x()*1000 + body->pos_y()*2000
                        + body->vel_x()*3000 + body->vel_y()*4000)) % 123456;
    result += Body_checksum;
  });

  return result;
}


#ifdef OPTION_RENDER
// Only for rendering: the sum of all body masses.
float max_mass = 0.0f;

void render_frame() {
  init_frame();

  allocator_handle->template device_do<Body>([&](Body* body) {
    draw_body(body->pos_x(), body->pos_y(), body->mass(), max_mass);

    allocator_handle->template device_do<Body>([&](Body* body2) {
      maybe_draw_line(body->pos_x(), body2->pos_x(),
                      body->pos_y(), body2->pos_y());
    });
  });

  show_frame();
}
#endif  // OPTION_RENDER


int main(int /*argc*/, char** /*argv*/) {
#ifdef OPTION_RENDER
  init_renderer();
#endif  // OPTION_RENDER

  // Create new allocator.
  allocator_handle = new AllocatorHandle<AllocatorT>(/*unified_memory=*/ true);
  AllocatorT* dev_ptr = allocator_handle->device_pointer();
  hipMemcpyToSymbol(HIP_SYMBOL(device_allocator), &dev_ptr, sizeof(AllocatorT*), 0,
                     hipMemcpyHostToDevice);

  // Allocate and create Body objects.
  allocator_handle->parallel_new<Body>(kNumBodies);

#ifdef OPTION_RENDER
  // Only for rendering: Calculate max_mass.
  // TODO: Provide an API for parallel reduce.
  max_mass = 0.0f;
  allocator_handle->template device_do<Body>([&](Body* body) {
    max_mass += body->mass();
  });
#endif  // OPTION_RENDER

  auto time_start = std::chrono::system_clock::now();

  for (int i = 0; i < kIterations; ++i) {
#ifndef NDEBUG
    // Print debug information.
    allocator_handle->DBG_print_state_stats();
#endif  // NDEBUG

    allocator_handle->parallel_do<Body, &Body::compute_force>();
    allocator_handle->parallel_do<Body, &Body::update>();
    allocator_handle->parallel_do<Body, &Body::initialize_merge>();
    allocator_handle->parallel_do<Body, &Body::prepare_merge>();
    allocator_handle->parallel_do<Body, &Body::update_merge>();
    allocator_handle->parallel_do<Body, &Body::delete_merged>();

#ifdef OPTION_RENDER
    render_frame();
#endif  // OPTION_RENDER
  }

  auto time_end = std::chrono::system_clock::now();
  auto elapsed = time_end - time_start;
  auto micros = std::chrono::duration_cast<std::chrono::microseconds>(elapsed)
      .count();

#ifndef NDEBUG
  printf("Checksum: %i\n", checksum());

  // TODO: Provide an API for counting objects.
  int num_remaining_bodies = 0;
  allocator_handle->template device_do<Body>([&](Body* body) {
    ++num_remaining_bodies;
  });

  printf("#bodies: %i\n", num_remaining_bodies);
#endif  // NDEBUG

  printf("%lu, %lu\n", micros, allocator_handle->DBG_get_enumeration_time());

#ifdef OPTION_RENDER
  close_renderer();
#endif  // OPTION_RENDER

  return 0;
}
