#include <chrono>

#include "nbody_dynasoar.h"

#include "configuration.h"
#include "rendering_dynasoar.h"

AllocatorHandle<AllocatorT>* allocator_handle;
__device__ AllocatorT* device_allocator;

float host_checksum;

__device__ Body::Body(float pos_x, float pos_y,
                      float vel_x, float vel_y, float mass)
    : pos_x_(pos_x), pos_y_(pos_y),
      vel_x_(vel_x), vel_y_(vel_y), mass_(mass) {}

__device__ void Body::compute_force() {
  /* TODO */
}

__device__ void Body::apply_force(Body* other) {
  // Update `other`.
  /* TODO */
}

__device__ void Body::update() {
  /* TODO */
}

void Body::add_checksum() {
  host_checksum += pos_x_ + pos_y_*2 + vel_x_*3 + vel_y_*4;
}

__device__ Body::Body(int idx) {
  hiprandState rand_state;
  hiprand_init(kSeed, idx, 0, &rand_state);

  pos_x_ = 2 * hiprand_uniform(&rand_state) - 1;
  pos_y_ = 2 * hiprand_uniform(&rand_state) - 1;
  vel_x_ = (hiprand_uniform(&rand_state) - 0.5) / 1000;
  vel_y_ = (hiprand_uniform(&rand_state) - 0.5) / 1000;
  mass_ = (hiprand_uniform(&rand_state)/2 + 0.5)* kMaxMass;
  force_x_ = 0.0f;
  force_y_ = 0.0f;
}

void step_simulation() {
  /* TODO */
}

void run_benchmark() {
  auto time_start = std::chrono::system_clock::now();

  for (int i = 0; i < kBenchmarkIterations; ++i) {
    step_simulation();
  }

  auto time_end = std::chrono::system_clock::now();
  auto elapsed = time_end - time_start;
  auto millis = std::chrono::duration_cast<std::chrono::milliseconds>(elapsed)
      .count();

  printf("Time: %lu ms\n", millis);
}

bool render_frame() {
  init_frame();

  allocator_handle->template device_do<Body>([&](Body* body){
    draw_body(body->pos_x(), body->pos_y(), body->mass());
  });

  return show_frame();
}

void run_interactive() {
  init_renderer();

  do {
    /* TODO */
  } while (render_frame());

  close_renderer();
}

int main(int argc, char** argv) {
  if (argc != 2) {
    printf("Usage: %s mode\n\nmode 0: Interactive mode\nmode 1: Benchmark\n",
           argv[0]);
    return 1;
  }

  int mode = atoi(argv[1]);

  allocator_handle = new AllocatorHandle<AllocatorT>(/*unified_memory=*/ true);
  AllocatorT *dev_ptr = allocator_handle->device_pointer();
  hipMemcpyToSymbol(HIP_SYMBOL(device_allocator), &dev_ptr, sizeof(AllocatorT*), 0,
                     hipMemcpyHostToDevice);
  
  // TODO: Initialize the objects

  if (mode == 0) {
    run_interactive();
  } else if (mode == 1) {
    run_benchmark();
    allocator_handle->template device_do<Body>(&Body::add_checksum);
    printf("Checksum: %i\n", static_cast<int>(host_checksum));
  } else {
    printf("Invalid mode.\n");
    return 1;
  }

  return 0;
}