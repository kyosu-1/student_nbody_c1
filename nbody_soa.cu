#include "hip/hip_runtime.h"
#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <new>

#include <hiprand/hiprand_kernel.h>

#include "configuration.h"
#include "cuda_helper.h"

// Arrays containing all Body objects on device.
__device__ float* dev_Body_pos_x;
__device__ float* dev_Body_pos_y;
__device__ float* dev_Body_vel_x;
__device__ float* dev_Body_vel_y;
__device__ float* dev_Body_mass;
__device__ float* dev_Body_force_x;
__device__ float* dev_Body_force_y;

float* host_Body_pos_x;
float* host_Body_pos_y;
float* host_Body_vel_x;
float* host_Body_vel_y;
float* host_Body_mass;
float* host_Body_force_x;
float* host_Body_force_y;


__device__ void new_Body(int id, float pos_x, float pos_y,
                         float vel_x, float vel_y, float mass) {
                           dev_Body_pos_x[id] = pos_x;
                           dev_Body_pos_y[id] = pos_y;
                           dev_Body_vel_x[id] = vel_x;
                           dev_Body_vel_y[id] = vel_y;
                           dev_Body_mass[id] = mass;
                           dev_Body_force_x[id] = 0;
                           dev_Body_force_y[id] = 0;
}

__device__ void compute_force(int id) {
  dev_Body_force_x[id] = 0;
  dev_Body_force_y[id] = 0;
  for (int i = 0; i < kNumBodies; ++i){
    if(id != i){
      float m1 = dev_Body_mass[id];
      float m2 = dev_Body_mass[i];
      float dx = dev_Body_pos_x[i] - dev_Body_pos_x[id];
      float dy = dev_Body_pos_y[i] - dev_Body_pos_y[id];
      float r = sqrt(dx * dx + dy * dy);
      float force = kGravityConstant * m1 * m2 / (r * r);
      dev_Body_force_x[id] += force * dx / r;
      dev_Body_force_y[id] += force * dy / r;
    }
  }
}

__device__ void update(float dt, int id) {
  dev_Body_vel_x[id] += dev_Body_force_x[id] * dt / dev_Body_mass[id];
  dev_Body_vel_y[id] += dev_Body_force_y[id] * dt / dev_Body_mass[id]t;
  dev_Body_pos_x[id] += dev_Body_vel_x[id] * dt;
  dev_Body_pos_y[id] += dev_Body_vel_y[id] * dt;
  if (abs(dev_Body_pos_x[id]) > 1) {
    dev_Body_vel_x[id] *= -1;
  }
  if (abs(dev_Body_pos_y[id]) > 1) {
    dev_Body_vel_y[i] *= -1;
  }
}


int Body_checksum(int id) {
  return host_Body_pos_x[id]*1000 + host_Body_pos_y[id]*2000
      + host_Body_vel_x[id]*3000 + host_Body_vel_y[id]*4000;
}


__global__ void kernel_initialize_bodies(float* pos_x, float* pos_y,
                                         float* vel_x, float* vel_y,
                                         float* mass, float* force_x,
                                         float* force_y) {
  dev_Body_pos_x = pos_x;
  dev_Body_pos_y = pos_y;
  dev_Body_vel_x = vel_x;
  dev_Body_vel_y = vel_y;
  dev_Body_mass = mass;
  dev_Body_force_x = force_x;
  dev_Body_force_y = force_y;

  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < kNumBodies; i += blockDim.x * gridDim.x) {
    // Initialize random state.
    hiprandState rand_state;
    hiprand_init(kSeed, i, 0, &rand_state);

    // Create new Body object.
    new_Body(/*id=*/ i,
             /*pos_x=*/ 2 * hiprand_uniform(&rand_state) - 1,
             /*pos_y=*/ 2 * hiprand_uniform(&rand_state) - 1,
             /*vel_x=*/ (hiprand_uniform(&rand_state) - 0.5) / 1000,
             /*vel_y=*/ (hiprand_uniform(&rand_state) - 0.5) / 1000,
             /*mass=*/ (hiprand_uniform(&rand_state)/2 + 0.5) * kMaxMass);
  }
}


__global__ void kernel_compute_force() {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < kNumBodies; i += blockDim.x * gridDim.x) {
        compute_force(i);
  }
}


__global__ void kernel_update() {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < kNumBodies; i += blockDim.x * gridDim.x) {
        update(kTimeInterval, i);
  }
}


// Compute one step of the simulation.
void step_simulation() {
  kernel_compute_force<<<128, 128>>>();
  gpuErrchk(hipDeviceSynchronize());

  kernel_update<<<128, 128>>>();
  gpuErrchk(hipDeviceSynchronize());
}


void run_benchmark() {
  auto time_start = std::chrono::system_clock::now();

  for (int i = 0; i < kBenchmarkIterations; ++i) {
    step_simulation();
  }

  auto time_end = std::chrono::system_clock::now();
  auto elapsed = time_end - time_start;
  auto millis = std::chrono::duration_cast<std::chrono::milliseconds>(elapsed)
      .count();

  printf("Time: %lu ms\n", millis);
}

int checksum() {
  int result = 0;

  for (int i = 0; i < kNumBodies; ++i) {
    result += Body_checksum(i);
    result %= 16785407;
  }

  return result;
}

int main(int argc, char** argv) {
  if (argc != 2) {
    printf("Usage: %s mode\n\nmode 1: Benchmark\n",
           argv[0]);
    return 1;
  }

  int mode = atoi(argv[1]);

  // Allocate and create Body objects.
  hipMallocManaged(&host_Body_pos_x, sizeof(float)*kNumBodies);
  hipMallocManaged(&host_Body_pos_y, sizeof(float)*kNumBodies);
  hipMallocManaged(&host_Body_vel_x, sizeof(float)*kNumBodies);
  hipMallocManaged(&host_Body_vel_y, sizeof(float)*kNumBodies);
  hipMallocManaged(&host_Body_mass, sizeof(float)*kNumBodies);
  hipMallocManaged(&host_Body_force_x, sizeof(float)*kNumBodies);
  hipMallocManaged(&host_Body_force_y, sizeof(float)*kNumBodies);

  kernel_initialize_bodies<<<128, 128>>>(host_Body_pos_x, host_Body_pos_y,
                                         host_Body_vel_x, host_Body_vel_y,
                                         host_Body_mass, host_Body_force_x,
                                         host_Body_force_y);
  gpuErrchk(hipDeviceSynchronize());

  if (mode == 1) {
    run_benchmark();
    printf("Checksum: %i\n", checksum());
  } else {
    printf("Invalid mode.\n");
    return 1;
  }

  hipFree(host_Body_pos_x);
  hipFree(host_Body_pos_y);
  hipFree(host_Body_vel_x);
  hipFree(host_Body_vel_y);
  hipFree(host_Body_mass);
  hipFree(host_Body_force_x);
  hipFree(host_Body_force_y);

  return 0;
}