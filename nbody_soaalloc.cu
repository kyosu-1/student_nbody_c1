#include "hip/hip_runtime.h"
#include <chrono>
#include <hiprand/hiprand_kernel.h>

#include "allocator/soa_allocator.h"
#include "allocator/soa_base.h"
#include "allocator/allocator_handle.h"

#include "configuration.h"
#include "rendering_soa.h"

#define OPTION_DRAW true

// Pre-declare all classes.
class Body;

using AllocatorT = SoaAllocator<64*64*64*64, Body>;

class Body : public SoaBase<AllocatorT> {
 public:
  using FieldTypes = std::tuple<float, float, float, float, float, float, float>;

  SoaField<Body, 0> pos_x_;
  SoaField<Body, 1> pos_y_;
  SoaField<Body, 2> vel_x_;
  SoaField<Body, 3> vel_y_;
  SoaField<Body, 4> mass_;
  SoaField<Body, 5> force_x_;
  SoaField<Body, 6> force_y_;


  __device__ Body(float pos_x, float pos_y, float vel_x, float vel_y, float mass);

  __device__ void compute_force();

  __device__ void apply_force(Body* other);

  __device__ void update();

  // Only for rendering purposes.
  __device__ void add_to_draw_array();
};

// Allocator handles.
AllocatorHandle<AllocatorT>* allocator_handle;
__device__ AllocatorT* device_allocator;


// Helper variables for rendering and checksum computation.
__device__ int draw_counter = 0;
__device__ float Body_pos_x[kNumBodies];
__device__ float Body_pos_y[kNumBodies];
__device__ float Body_vel_x[kNumBodies];
__device__ float Body_vel_y[kNumBodies];
__device__ float Body_mass[kNumBodies];
float host_Body_pos_x[kNumBodies];
float host_Body_pos_y[kNumBodies];
float host_Body_vel_x[kNumBodies];
float host_Body_vel_y[kNumBodies];
float host_Body_mass[kNumBodies];

__device__ Body::Body(float pos_x, float pos_y,
                      float vel_x, float vel_y, float mass)
    : pos_x_(pos_x), pos_y_(pos_y),
      vel_x_(vel_x), vel_y_(vel_y), mass_(mass) {}


__device__ void Body::compute_force() {
  force_x_ = 0;
  force_y_ = 0;
  device_allocator->template device_do<Body>(&Body::apply_force, this);
  /*for (int i = 0; i < kNumBodies; ++i){
    this.apply_force(dev_bodies + i)*/
}



__device__ void Body::apply_force(Body* other) {
  // Update `other`.
  if(this != other){
    float m1 = this->mass_;
    float dx = this->pos_x_ - other->pos_x_;
    float dy = this->pos_y_ - other->pos_y_;
    float r = sqrt(dx * dx + dy * dy);
    other->force_x_ += kGravityConstant * m1 * other->mass_ / (r * r * r) * dx;
    other->force_y_ += kGravityConstant * m1 * other->mass_ / (r * r * r) * dy;
  }
}


__device__ void Body::update() {
  vel_x_ += force_x_ / mass_ * kTimeInterval;
  vel_y_ += force_y_ / mass_ * kTimeInterval;
  pos_x_ += vel_x_ * kTimeInterval;
  pos_y_ += vel_y_ * kTimeInterval;
  if (abs(pos_x_) > 1) {
    vel_x_ = - vel_x_;
  }
  if (abs(pos_y_) > 1) {
    vel_y_ = - vel_y_;
  }
}


__device__ void Body::add_to_draw_array() {
  int idx = atomicAdd(&draw_counter, 1);
  Body_pos_x[idx] = pos_x_;
  Body_pos_y[idx] = pos_y_;
  Body_vel_x[idx] = vel_x_;
  Body_vel_y[idx] = vel_y_;
  Body_mass[idx] = mass_;
}


__global__ void kernel_initialize_bodies() {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  hiprandState rand_state;
  hiprand_init(kSeed, tid, 0, &rand_state);

  for (int i = tid; i < kNumBodies; i += blockDim.x * gridDim.x) {


    // Initialize random state.
    hiprandState rand_state;
    hiprand_init(kSeed, i, 0, &rand_state);

    // Create new Body object with placement-new.

    device_allocator->make_new<Body>(/*pos_x=*/ 2 * hiprand_uniform(&rand_state) - 1,
                             /*pos_y=*/ 2 * hiprand_uniform(&rand_state) - 1,
                             /*vel_x=*/ (hiprand_uniform(&rand_state) - 0.5) / 1000,
                             /*vel_y=*/ (hiprand_uniform(&rand_state) - 0.5) / 1000,
                             /*mass=*/ (hiprand_uniform(&rand_state)/2 + 0.5)
                                           * kMaxMass);
  }
}


__global__ void kernel_reset_draw_counters() {
  draw_counter = 0;
}


void transfer_data() {
  // Extract data from SoaAlloc data structure.
  kernel_reset_draw_counters<<<1, 1>>>();
  gpuErrchk(hipDeviceSynchronize());
  allocator_handle->parallel_do<Body, &Body::add_to_draw_array>();
  gpuErrchk(hipDeviceSynchronize());

  // Copy data to host.
  hipMemcpyFromSymbol(host_Body_pos_x, HIP_SYMBOL(Body_pos_x),
                       sizeof(float)*kNumBodies, 0, hipMemcpyDeviceToHost);
  hipMemcpyFromSymbol(host_Body_pos_y, HIP_SYMBOL(Body_pos_y),
  	                   sizeof(float)*kNumBodies, 0, hipMemcpyDeviceToHost);
  hipMemcpyFromSymbol(host_Body_vel_x, HIP_SYMBOL(Body_vel_x),
  	                   sizeof(float)*kNumBodies, 0, hipMemcpyDeviceToHost);
  hipMemcpyFromSymbol(host_Body_vel_y, HIP_SYMBOL(Body_vel_y),
  	                   sizeof(float)*kNumBodies, 0, hipMemcpyDeviceToHost);
  hipMemcpyFromSymbol(host_Body_mass, HIP_SYMBOL(Body_mass), sizeof(float)*kNumBodies, 0,
                       hipMemcpyDeviceToHost);
}


int checksum() {
  transfer_data();
  int result = 0;

  for (int i = 0; i < kNumBodies; ++i) {
  	int Body_checksum = static_cast<int>((host_Body_pos_x[i]*1000 + host_Body_pos_y[i]*2000
                        + host_Body_vel_x[i]*3000 + host_Body_vel_y[i]*4000)) % 123456;
    result += Body_checksum;
  }

  return result;
}


void run_interactive() {
  init_renderer();

  while (true) {
    allocator_handle->parallel_do<Body, &Body::compute_force>();
    allocator_handle->parallel_do<Body, &Body::update>();

    // Transfer and render.
    transfer_data();
    draw(host_Body_pos_x, host_Body_pos_y, host_Body_mass);
  }

  close_renderer();
}


void run_benchmark() {
  auto time_start = std::chrono::system_clock::now();

  for (int i = 0; i < kBenchmarkIterations; ++i) {
    allocator_handle->parallel_do<Body, &Body::compute_force>();
    allocator_handle->parallel_do<Body, &Body::update>();
  }

  auto time_end = std::chrono::system_clock::now();
  auto elapsed = time_end - time_start;
  auto millis = std::chrono::duration_cast<std::chrono::milliseconds>(elapsed)
      .count();

  printf("Time: %lu ms\n", millis);
}


int main(int argc, char** argv) {
  if (argc != 2) {
    printf("Usage: %s mode\n\nmode 0: Interactive mode\nmode 1: Benchmark\n",
           argv[0]);
    return 1;
  }

  int mode = atoi(argv[1]);

  AllocatorT::DBG_print_stats();

  // Create new allocator.
  allocator_handle = new AllocatorHandle<AllocatorT>();
  AllocatorT* dev_ptr = allocator_handle->device_pointer();
  hipMemcpyToSymbol(HIP_SYMBOL(device_allocator), &dev_ptr, sizeof(AllocatorT*), 0,
                     hipMemcpyHostToDevice);

  // Allocate and create Body objects.
  kernel_initialize_bodies<<<128, 128>>>();
  gpuErrchk(hipDeviceSynchronize());

  if (mode == 0) {
    run_interactive();
  } else if (mode == 1) {
    run_benchmark();
    printf("Checksum: %i\n", checksum());
  } else {
    printf("Invalid mode.\n");
    return 1;
  }

  return 0;
}
